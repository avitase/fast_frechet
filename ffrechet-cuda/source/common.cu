#include <stdexcept>
#include <string>

#include "ffrechet-cuda/ffrechet-cuda.hpp"

namespace fast_frechet
{
namespace cuda
{
void cuda_check(hipError_t code)
{
    if (code != hipSuccess)
    {
        throw std::runtime_error{"CUDA error: " + std::string{hipGetErrorString(code)}};
    }
}

void cuda_check()
{
    cuda_check(hipGetLastError());
}
} // namespace cuda
} // namespace fast_frechet
