#include "hip/hip_runtime.h"
#include <vector>

#include "ffrechet-cuda/common.hpp"
#include "ffrechet-cuda/ffrechet-cuda.hpp"

__device__ static float metric(float dx, float dy)
{
    return hypot(dx, dy);
}

__global__ static void kernel(const float* px,
                              const float* py,
                              const unsigned* P,
                              const unsigned N,
                              const float* qx,
                              const float* qy,
                              unsigned Q,
                              float* buffer,
                              float* res)
{
    for (auto idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += gridDim.x * blockDim.x)
    {
        float acc = 0.F;
        for (unsigned j = 0; j < Q; j++)
        {
            acc = max(acc, metric(px[idx] - qx[j], py[idx] - qy[j]));
            buffer[j * N + idx] = acc;
        }

        for (unsigned i = 1; i < P[idx]; i++)
        {
            for (unsigned j = Q - 1; j > 0; j--)
            {
                buffer[j * N + idx] = min(buffer[j * N + idx], buffer[(j - 1) * N + idx]);
            }
            buffer[idx] =
                max(buffer[idx], metric(px[i * N + idx] - qx[0], py[i * N + idx] - qy[0]));

            for (unsigned j = 1; j < Q; j++)
            {
                buffer[j * N + idx] = max(min(buffer[j * N + idx], buffer[(j - 1) * N + idx]),
                                          metric(px[i * N + idx] - qx[j], py[i * N + idx] - qy[j]));
            }
        }

        res[idx] = buffer[(Q - 1) * N + idx];
    }
}

namespace
{
template <typename T>
[[nodiscard]] T max_element(const T* v, std::size_t N)
{
    T max = v[0];

    for (std::size_t i = 1; i < N; i++)
    {
        max = std::max(max, v[i]);
    }

    return max;
}
} // namespace

void cuda_frechet_distance(const float* const* px,
                           const float* const* py,
                           const unsigned* P,
                           const unsigned N,
                           const float* qx,
                           const float* qy,
                           unsigned Q,
                           float* d,
                           CUDAKernelConfig cfg)
{
    using ::fast_frechet::cuda::cuda_check;

    const auto P_MAX = ::max_element(P, N);

    std::vector<float> buffer(static_cast<std::size_t>(N) * Q);
    std::vector<float> px_flat(static_cast<std::size_t>(N) * P_MAX);
    std::vector<float> py_flat(static_cast<std::size_t>(N) * P_MAX);

    for (std::size_t i = 0; i < N; i++)
    {
        for (std::size_t j = 0; j < P_MAX; j++)
        {
            const auto k = std::min(j, static_cast<std::size_t>(P[i]) - 1U);
            px_flat[j * N + i] = px[i][k];
            py_flat[j * N + i] = py[i][k];
        }
    }

    float* px_d;
    cuda_check(hipMalloc(&px_d, px_flat.size() * sizeof(float)));
    cuda_check(
        hipMemcpy(px_d, px_flat.data(), px_flat.size() * sizeof(float), hipMemcpyHostToDevice));

    float* py_d;
    cuda_check(hipMalloc(&py_d, py_flat.size() * sizeof(float)));
    cuda_check(
        hipMemcpy(py_d, py_flat.data(), py_flat.size() * sizeof(float), hipMemcpyHostToDevice));

    unsigned* P_d;
    cuda_check(hipMalloc(&P_d, N * sizeof(unsigned)));
    cuda_check(hipMemcpy(P_d, P, N * sizeof(unsigned), hipMemcpyHostToDevice));

    float* qx_d;
    cuda_check(hipMalloc(&qx_d, Q * sizeof(float)));
    cuda_check(hipMemcpy(qx_d, qx, Q * sizeof(float), hipMemcpyHostToDevice));

    float* qy_d;
    cuda_check(hipMalloc(&qy_d, Q * sizeof(float)));
    cuda_check(hipMemcpy(qy_d, qy, Q * sizeof(float), hipMemcpyHostToDevice));

    float* buffer_d;
    cuda_check(hipMalloc(&buffer_d, Q * N * sizeof(float)));

    float* res_d;
    cuda_check(hipMalloc(&res_d, N * sizeof(float)));

    kernel<<<cfg.grid_size, cfg.block_size>>>(px_d, py_d, P_d, N, qx_d, qy_d, Q, buffer_d, res_d);
    cuda_check();
    cuda_check(hipDeviceSynchronize());

    cuda_check(hipMemcpy(d, res_d, N * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(px_d);
    hipFree(py_d);
    hipFree(qx_d);
    hipFree(qy_d);
    hipFree(buffer_d);
    hipFree(res_d);
}
